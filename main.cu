#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cstdio>

#include "common.h"
#include "transformer.h"
//#include "cublas_utils.h"

int main(void) {
    hipblasHandle_t handle = NULL;
    hipblasCreate(&handle);
    int m = 4;
    int k = 8;
    int n = 1;

    float* host_A;
    float* host_B;
    float* host_C;

    Fp16* host_A_Hi;
    Fp16* host_A_Lo;
    Fp16* host_B_Hi;
    Fp16* host_B_Lo;
    float* host_shift_A;
    float* host_shift_B;

    host_A = (float*)malloc(sizeof(float) * m * k);
    host_B = (float*)malloc(sizeof(float) * n * k);
    host_C = (float*)malloc(sizeof(float) * m * n);

    host_A_Hi = (Fp16*)malloc(sizeof(Fp16) * m * k);
    host_A_Lo = (Fp16*)malloc(sizeof(Fp16) * m * k);
    host_B_Hi = (Fp16*)malloc(sizeof(Fp16) * k * n);
    host_B_Lo = (Fp16*)malloc(sizeof(Fp16) * k * n);

    host_shift_A = (float*)malloc(sizeof(float) * m * k);
    host_shift_B = (float*)malloc(sizeof(float) * n * k);

    for (int i = 0; i < m; i++) {
        for (int j = 0; j < k; j++) {
            Fp32 m;
            m.ui = 0x45177fe6;
            host_A[i*k + j] = m.fp;
            Fp16 hi, lo;
            unsigned int Hi_shift = toFp32_F(m.fp, &hi, &lo);
            host_A_Hi[i*k + j] = hi;
            host_A_Lo[i*k + j] = lo;
            if (Hi_shift >= 0) {
                host_shift_A[i*k + j] = pow(2.f, Hi_shift);
            } else {
                host_shift_A[i*k + j] = pow(0.5f, -Hi_shift);
            }
            
        }
    }

    for (int i = 0; i < k; i++) {
        for (int j = 0; j < n; j++) {
            Fp32 m;
            m.ui = 0x3f7a574e;
            host_B[i*n + j] = m.fp;
            Fp16 hi, lo;
            unsigned int Hi_shift = toFp32_F(m.fp, &hi, &lo);
            host_B_Hi[i*k + j] = hi;
            host_B_Lo[i*k + j] = lo;
            if (Hi_shift >= 0) {
                host_shift_B[i*k + j] = pow(2.f, Hi_shift);
            } else {
                host_shift_B[i*k + j] = pow(0.5f, -Hi_shift);
            }
        }
    }

    float* dev_A;
    float* dev_B;
    float* dev_C;

    hipMalloc((void**) &dev_A, m * k * sizeof(float));
    hipMalloc((void**) &dev_B, n * k * sizeof(float));
    hipMalloc((void**) &dev_C, m * n * sizeof(float));

    hipMemcpy(dev_A, host_A, m * k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_B, host_B, n * k * sizeof(float), hipMemcpyHostToDevice);

    const float alpha = 1.0;
    const float beta = 0.0;

    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, dev_A, m, dev_B, k, &beta, dev_C, m);

    hipMemcpy(host_C, dev_C, n * m * sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            printf("%f ", host_C[i*n + j]);
        }
        printf("\n");
    }

    hipMemset(dev_C, 0.f, m * n);

    fp16* dev_A_Hi;
    fp16* dev_A_Lo;
    fp16* dev_B_Hi;
    fp16* dev_B_Lo;

    float* dev_shift_A;
    float* dev_shift_B;

    hipMalloc((void**)&dev_A_Hi, sizeof(fp16) * m * k);
    hipMalloc((void**)&dev_A_Lo, sizeof(fp16) * m * k);
    hipMalloc((void**)&dev_B_Hi, sizeof(fp16) * n * k);
    hipMalloc((void**)&dev_B_Lo, sizeof(fp16) * n * k);
    hipMalloc((void**)&dev_shift_A, sizeof(float) * m * k);
    hipMalloc((void**)&dev_shift_B, sizeof(float) * n * k);

    hipMemcpy(&dev_A_Hi, host_A_Hi, sizeof(fp16) * m * k, hipMemcpyDeviceToHost);
    hipMemcpy(&dev_A_Lo, host_A_Lo, sizeof(fp16) * m * k, hipMemcpyDeviceToHost);
    hipMemcpy(&dev_B_Hi, host_B_Hi, sizeof(fp16) * n * k, hipMemcpyDeviceToHost);
    hipMemcpy(&dev_B_Lo, host_B_Lo, sizeof(fp16) * n * k, hipMemcpyDeviceToHost);

    hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, dev_A_Hi, HIP_R_16F, m, dev_B_Hi, HIP_R_16F, k, &beta, dev_C, HIP_R_32F, m, HIP_R_16F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, dev_A_Hi, HIP_R_16F, m, dev_B_Lo, HIP_R_16F, k, &beta, dev_C, HIP_R_32F, m, HIP_R_16F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, dev_A_Lo, HIP_R_16F, m, dev_B_Hi, HIP_R_16F, k, &beta, dev_C, HIP_R_32F, m, HIP_R_16F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    
    //print_matrix(m, n, host_C, n);
    //hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, (void*)dev_A_Hi, (void*)dev_A_Hi, HIP_R_16F, 8, (void*)dev_B_Hi, HIP_R_16F, 8, (void*)dev_B_Hi, (void*)dev_C, HIP_R_16F, 8, HIP_R_16F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    return 0;
}