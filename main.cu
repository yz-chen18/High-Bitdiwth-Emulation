#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cstdio>

#include "common.h"
#include "transformer.h"
//#include "cublas_utils.h"

int main(void) {
    hipblasHandle_t handle = NULL;
    hipblasCreate(&handle);
    int m = 8;
    int k = 128;
    int n = 128;
    int s = 8;

    float* host_A;
    float* host_B;
    float* host_C;

    Fp16* host_A_Hi;
    Fp16* host_A_Lo;
    Fp16* host_B_Hi;
    Fp16* host_B_Lo;

    host_A = (float*)malloc(sizeof(float) * m * k);
    host_B = (float*)malloc(sizeof(float) * n * k);
    host_C = (float*)malloc(sizeof(float) * m * n);

    host_A_Hi = (Fp16*)malloc(sizeof(Fp16) * m * k);
    host_A_Lo = (Fp16*)malloc(sizeof(Fp16) * m * k);
    host_B_Hi = (Fp16*)malloc(sizeof(Fp16) * k * n);
    host_B_Lo = (Fp16*)malloc(sizeof(Fp16) * k * n);

    for (int i = 0; i < m; i++) {
        for (int j = 0; j < k; j++) {
            Fp32 m;
            //m.ui = 0x45177fe6;
            host_A[i*k + j] = i*k + j;
        }
    }

    for (int i = 0; i < k; i++) {
        for (int j = 0; j < n; j++) {
            Fp32 m;
            //m.ui = 0x3f7a574e;
            host_B[i*n + j] = i*n + j;
        }
    }

    float* dev_A;
    float* dev_B;
    float* dev_C;

    hipMalloc((void**) &dev_A, m * k * sizeof(float));
    hipMalloc((void**) &dev_B, n * k * sizeof(float));
    hipMalloc((void**) &dev_C, m * n * sizeof(float));

    hipMemcpy(dev_A, host_A, m * k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_B, host_B, n * k * sizeof(float), hipMemcpyHostToDevice);

    const float alpha = 1.0;
    const float beta = 1.0;

    hipMemset(dev_C, 0, sizeof(float) * m * n);
    hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, dev_A, HIP_R_32F, m, dev_B, HIP_R_32F, k, &beta, dev_C, HIP_R_32F, m, HIPBLAS_COMPUTE_32F, HIPBLAS_GEMM_DEFAULT);

    printf("CUDA\n");
    hipMemcpy(host_C, dev_C, n * m * sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            printf("%e ", host_C[i*n + j]);
        }
        printf("\n");
    }

    matrix_stride_transpose(k, n, s, host_B);
    hipMemcpy(dev_B, host_B, n * k * sizeof(float), hipMemcpyHostToDevice);

    hipMemset(dev_C, 0, sizeof(float) * m * n);
    for (int i = 0; i < k / s; i++)
        hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, s, &alpha, dev_A + i*m, HIP_R_32F, m*k/s, dev_B + i*s, HIP_R_32F, k, &beta, dev_C, HIP_R_32F, m, HIPBLAS_COMPUTE_32F, HIPBLAS_GEMM_DEFAULT);    
    //hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, s, &alpha, dev_A + m, HIP_R_32F, m*2, dev_B + 4, HIP_R_32F, k, &beta, dev_C, HIP_R_32F, m, HIPBLAS_COMPUTE_32F, HIPBLAS_GEMM_DEFAULT); 
    //hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, 4, &alpha, dev_A, HIP_R_32F, m*2, dev_B + 8, HIP_R_32F, k, &beta, dev_C, HIP_R_32F, m, HIPBLAS_COMPUTE_32F, HIPBLAS_GEMM_DEFAULT); 
    //hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, 4, &alpha, dev_A, HIP_R_32F, m*2, dev_B + 12, HIP_R_32F, k, &beta, dev_C, HIP_R_32F, m, HIPBLAS_COMPUTE_32F, HIPBLAS_GEMM_DEFAULT); 
    
    printf("TENSOR\n");
    hipMemcpy(host_C, dev_C, n * m * sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            printf("%e ", host_C[i*n + j]);
        }
        printf("\n");
    }
    return 0;
}