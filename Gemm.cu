#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "common.h"
#include "error.h"
#include "cublas_utils.h"

void Gemm(int m, int n, int k, const Fp16* host_A_Hi, const Fp16* host_A_Lo, const Fp16* host_B_Hi, const Fp16* host_B_Lo, 
  const Fp16* host_C, Fp32* host_D) {
    if ((m % 4 != 0) || (k % 8 != 0)) error("Gemm", "m must be a multiple of 4, k must be a multiple of 8");
    hipblasHandle_t handle = NULL;
    CUBLAS_CHECK(hipblasCreate(&handle));
    CUBLAS_CHECK(hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH));

    Fp16* dev_A_Hi;
    Fp16* dev_A_Lo;
    Fp16* dev_B_Hi;
    Fp16* dev_B_Lo;
    Fp16* dev_C;

    const float alpha = 1.0;
    const float beta = 0.0;

    hipMalloc((void**) &dev_A_Hi, m * k * sizeof(Fp16));
    hipMalloc((void**) &dev_A_Lo, m * k * sizeof(Fp16));
    hipMalloc((void**) &dev_B_Hi, n * k * sizeof(Fp16));
    hipMalloc((void**) &dev_B_Lo, n * k * sizeof(Fp16));
    hipMalloc((void**) &dev_C, m * n * sizeof(Fp16));

    hipMemcpy(dev_A_Hi, host_A_Hi, hipMemcpyHostToDevice);
    hipMemcpy(dev_A_Lo, host_A_Lo, hipMemcpyHostToDevice);
    hipMemcpy(dev_B_Hi, host_B_Hi, hipMemcpyHostToDevice);
    hipMemcpy(dev_B_Lo, host_B_Lo, hipMemcpyHostToDevice);
    hipMemcpy(dev_C, host_C, hipMemcpyHostToDevice);

    hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, dev_A_Hi, HIP_R_16F, m, dev_B_Hi, HIP_R_16F, k, &beta, dev_C, HIP_R_16F, m, HIP_R_16F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
}